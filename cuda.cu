#include "hip/hip_runtime.h"
#include <cstdio>
#include <hip/hip_runtime.h>

// cuRAND
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include "grid.h"

extern "C"

dim3 blocks(8, 8, 8);
dim3 threads(8, 8, 8);

__device__
int getIndex(int xSize, int ySize, int zSize, int i, int j, int k)
{
    return (i * ySize + j) * zSize + k;
}

__global__
void kernelInitRandomStates(hiprandState *randomStates, int xSize, int ySize, int zSize)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;
    int offsetx = gridDim.x * blockDim.x;
    int offsety = gridDim.y * blockDim.y;
    int offsetz = gridDim.z * blockDim.z;

    int index = getIndex(offsetx, offsety, offsetz, idx, idy, idz);
    hiprand_init(clock64(), index, 0, &(randomStates[index]));
}

__global__
void kernelInitGrid(
    BYTE *data, hiprandState *randomStates,
    int xSize, int ySize, int zSize
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;
    int offsetx = gridDim.x * blockDim.x;
    int offsety = gridDim.y * blockDim.y;
    int offsetz = gridDim.z * blockDim.z;

    hiprandState *rndState = &(randomStates[getIndex(offsetx, offsety, offsetz, idx, idy, idz)]);

    for (int i = idx; i < xSize; i += offsetx)
    {
        for (int j = idy; j < ySize; j += offsety)
        {
            for (int k = idz; k < zSize; k += offsetz)
            {
                BYTE value = hiprand_uniform(rndState) < 0.5 ? 0 : 2;
                data[getIndex(xSize, ySize, zSize, i, j, k)] = value;
            }
        }
    }
}

__device__
double gridDistantion(int i, int j, int k, int x, int y, int z)
{
    return sqrt(double((i - x) * (i - x) + (j - y) * (j - y) + (k - z) * (k - z)));
}

__device__
double gridInteractionPotential(double interactionEnergy, double r)
{
    return interactionEnergy / (r * r);
}

__global__
void kernelAlgorithm(
    BYTE *data, hiprandState *randomStates,
    DimensionEnum dimension, int xSize, int ySize, int zSize,
    int interactionEnergy, double externalField, int interactionRadius, double temperature,
    bool even
)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int idz = blockIdx.z * blockDim.z + threadIdx.z;
    int offsetx = gridDim.x * blockDim.x;
    int offsety = gridDim.y * blockDim.y;
    int offsetz = gridDim.z * blockDim.z;

    hiprandState *rndState = &(randomStates[getIndex(offsetx, offsety, offsetz, idx, idy, idz)]);

    for (int i = idx; i < xSize; i += offsetx)
    {
        for (int j = idy; j < ySize; j += offsety)
        {
            for (int k = idz; k < zSize; k += offsetz)
            {
                int index = getIndex(xSize, ySize, zSize, i, j, k);

                if (even && index % 2 != 0) continue;
                if (!even && index % 2 == 0) continue;

                int radiusX = interactionRadius;
                int radiusY = interactionRadius;
                int radiusZ = interactionRadius;

                switch (dimension)
                {
                    case DIM_1: radiusY = radiusZ = 0; break;
                    case DIM_2: radiusZ = 0; break;
                    case DIM_3: break;
                }

                double gridSpinEnergy = externalField;
                for (int x = i - radiusX; x <= i + radiusX; x++)
                {
                    for (int y = j - radiusY; y <= j + radiusY; y++)
                    {
                        for (int z = k - radiusZ; z <= k + radiusZ; z++)
                        {
                            int xx = (xSize + x) % xSize;
                            int yy = (ySize + y) % ySize;
                            int zz = (zSize + z) % zSize;

                            if (xx == i && yy == j && zz == k) continue;

                            double dist = gridDistantion(i, j, k, xx, yy, zz);

                            if (dist <= interactionRadius) {
                                gridSpinEnergy += (data[getIndex(xSize, ySize, zSize, xx, yy, zz)] - 1)
                                                  * gridInteractionPotential(interactionEnergy, dist);
                            }
                        }
                    }
                }

                double expValue = exp(gridSpinEnergy / temperature);
                double probplus  = expValue;
                double probminus = 1.0 / expValue;
                double probability = probplus / (probplus + probminus);

                if (hiprand_uniform(rndState) > probability)
                {
                    data[index] = 0;
                }
                else
                {
                    data[index] = 2;
                }
            }
        }
    }
}

void cudaInitGrid(Grid *g)
{
    int x = blocks.x * threads.x;
    int y = blocks.y * threads.y;
    int z = blocks.z * threads.z;
    hipMalloc((void **)& (g->randomStates), sizeof(hiprandState) * x * y * z);

    int dataSize = g->xSize * g->ySize * g->zSize;
    hipMalloc((void **)& (g->deviceMatrix), sizeof(BYTE) * dataSize);
    kernelInitRandomStates<<<blocks, threads>>>(g->randomStates, g->xSize, g->ySize, g->zSize);
    kernelInitGrid<<<blocks, threads>>>(g->deviceMatrix, g->randomStates, g->xSize, g->ySize, g->zSize);
}

void cudaFreeGrid(Grid *g)
{
    if (g->randomStates != NULL) hipFree(g->randomStates);
    if (g->deviceMatrix != NULL) hipFree(g->deviceMatrix);
}

void cudaUpdateTempMatrix(Grid *g)
{
    int dataSize = g->xSize * g->ySize * g->zSize;
    hipMemcpy(g->hostMatrix, g->deviceMatrix, sizeof(BYTE) * dataSize, hipMemcpyDeviceToHost);
}

void cudaAlgorithmStep(Grid *g, int algorithmSteps)
{
    for (int i = 0; i < algorithmSteps; i++)
    {
        kernelAlgorithm<<<blocks, threads>>>(
            g->deviceMatrix, g->randomStates,
            g->dimension, g->xSize, g->ySize, g->zSize,
            g->interactionEnergy, g->externalField, g->interactionRadius, g->temperature, false
        );

        kernelAlgorithm<<<blocks, threads>>>(
            g->deviceMatrix, g->randomStates,
            g->dimension, g->xSize, g->ySize, g->zSize,
            g->interactionEnergy, g->externalField, g->interactionRadius, g->temperature, true
        );
    }
}
